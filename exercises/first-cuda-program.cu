
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>

__global__ void set(int *A, int N) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
        A[idx] = idx;
}

int main(void)
{
    const int N = 128;

    int *d_A; // Device memory
    int *h_A; // Host memory

    h_A = (int*) malloc(N*sizeof(int));

    // Allocate linear memory on the device
    hipMalloc(&d_A, N*sizeof(int));
    // For 2D or 3D memory, use cudaMallocPitch or cudaMalloc3D respectively
    
    // Number of blocks, number of threads per block
    set<<<2, N / 2>>>(d_A, N);

    hipMemcpy(h_A, d_A, N*sizeof(int), hipMemcpyDeviceToHost);

    // Check errors
    hipError_t syncErr = hipDeviceSynchronize();
    hipError_t asyncErr = hipGetLastError();
    if (syncErr != hipSuccess)
        printf("Sync error: %s\n", hipGetErrorString(syncErr));
    if (asyncErr != hipSuccess)
        printf("Async error: %s\n", hipGetErrorString(asyncErr));

    // Output
    for (int i = 0; i < N; i++)
        printf("%i ", h_A[i]);
    printf("\n");

    free(h_A);

    // Free device memory
    hipFree(d_A);

    return 0;
}
