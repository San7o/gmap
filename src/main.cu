
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel function
__global__ void helloFromGPU() {
    printf("Hello World from GPU thread %d!\n", threadIdx.x);
}

int main() {
    // Launch the kernel with 1 block and 10 threads
    helloFromGPU<<<1, 10>>>();

    // Wait for GPU to finish before accessing on host
    hipError_t syncErr = hipDeviceSynchronize();
    hipError_t asyncErr = hipGetLastError();

    if (syncErr != hipSuccess) {
        printf("Sync error: %s\n", hipGetErrorString(syncErr));
    }
    if (asyncErr != hipSuccess) {
        printf("Async error: %s\n", hipGetErrorString(asyncErr));
    }

    printf("Hello World from CPU!\n");

    return 0;
}
